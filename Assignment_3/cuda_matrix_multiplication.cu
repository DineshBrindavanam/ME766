#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

// CUDA kernel for matrix multiplication
__global__ void matrixMultiplication(float* A, float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float value = 0;
        for (int k = 0; k < N; ++k) {
            value += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = value;
    }
}

// Helper function to initialize matrix with random values
void matrixInitialize(float* matrix, int N) {
    for (int i = 0; i < N * N; ++i) {
        matrix[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

int main() {
    for (int N = 100; N <= 10000; N += 100) { // Adjust step size as needed
        int size = N * N * sizeof(float);

        // Allocate memory on host
        float *h_A, *h_B, *h_C;
        h_A = (float*)malloc(size);
        h_B = (float*)malloc(size);
        h_C = (float*)malloc(size);

        // Initialize matrices A and B
        matrixInitialize(h_A, N);
        matrixInitialize(h_B, N);

        // Allocate memory on device
        float *d_A, *d_B, *d_C;
        hipMalloc((void**)&d_A, size);
        hipMalloc((void**)&d_B, size);
        hipMalloc((void**)&d_C, size);

        // Copy data from host to device
        hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

        // Define thread block and grid dimensions
        dim3 threadsPerBlock(16, 16);
        dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                           (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

        // Measure time
        auto start = std::chrono::high_resolution_clock::now();

        // Launch kernel
        matrixMultiplication<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
        hipDeviceSynchronize();

        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<float, std::milli> duration = end - start;
        std::cout << "Matrix size N=" << N << ", Time: " << duration.count() << " ms" << std::endl;

        // Copy result back to host
        hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

        // Free memory
        free(h_A); free(h_B); free(h_C);
        hipFree(d_A); hipFree(d_B); hipFree(d_C);
    }

    return 0;
}
